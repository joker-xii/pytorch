#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/core/TensorAccessor.h>
#include <ATen/cuda/HIPContext.h>

#include <assert.h>

using namespace at;

__global__ void test_tensor_packed_accessor_kernel(
    PackedTensorAccessor<float, 1, RestrictPtrTraits> resa,
    PackedTensorAccessor<float, 2, RestrictPtrTraits> t1a,
    PackedTensorAccessor<float, 1, RestrictPtrTraits> t2a) {
  for (int64_t i = 0; i < resa.size(0); i++) {
    float val = 0.0f;
    for (int64_t j = 0; j < t1a.size(1); j++) {
      val += t1a[i][j] * t2a[j];
    }
    resa[i] = val;
  }
}

__global__ void test_std_vector_tensor(
    std::vector<torch::Tensor> v_t1a) {
  double total = 0;
  for (int64_t i = 0; i < v_t1a.size(); i++) {
    auto accessor = v_t1a[i];
    for (int64_t j = 0; j < accessor.size(0); j++) {
      total += accessor[j];
    }
  }
  v_t1a[0][0] = total;
}

// test PackedTensorAccessor and Tensor.packed_accessor
TEST(PackedtensoraccessorTest, PackedtensoraccessorTestCUDA) {
  if (!at::cuda::is_available()) return;
  manual_seed(123);

  Tensor t1 = rand({4, 4}, CUDA(kFloat));
  Tensor t2 = rand({4}, CUDA(kFloat));
  Tensor res = empty({4}, CUDA(kFloat));

  auto t1a = t1.packed_accessor<float, 2, RestrictPtrTraits>();
  auto t2a = t2.packed_accessor<float, 1, RestrictPtrTraits>();
  auto resa = res.packed_accessor<float, 1, RestrictPtrTraits>();

  auto stream = at::cuda::getCurrentCUDAStream();

  test_tensor_packed_accessor_kernel<<<1, 1, 0, stream>>>(resa, t1a, t2a);
  hipError_t err = hipDeviceSynchronize();
  bool isEQ = err == hipSuccess;
  ASSERT_TRUE(isEQ);

  auto expected = mv(t1, t2);

  ASSERT_TRUE(res.allclose(expected));
}

TEST(PackedtensoraccessorTest, PackedtensoraccessorTestStdVector) {
  if (!at::cuda::is_available()) throw 42;

  Tensor t1 = empty({4}, CUDA(kFloat));

  // auto t1a = t1.packed_accessor<float, 1, RestrictPtrTraits>();

  std::vector<torch::Tensor> vec = {t1};

  auto stream = at::cuda::getCurrentCUDAStream();

  test_tensor_packed_accessor_kernel<<<1, 1, 0, stream>>>(vec);
  hipError_t err = hipDeviceSynchronize();
  bool isEQ = err == hipSuccess;
  ASSERT_TRUE(isEQ);
}
